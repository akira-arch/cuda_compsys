#include "hip/hip_runtime.h"
/* https://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf */

#include "./cuda_class.hpp"
#include "cuda_worker.hpp"

 // Get information about the worker
void ComputeUnifiedDeviceArchitectureWorker::get_info() {
    std::cout << "Sizee: " << this->_size << std::endl;
    std::cout << "<text>: " << this->_copies.size() << std::endl;
    std::cout << "GPU Blocks: " << this->_dim->grid << std::endl;
};

// Create a new subscriber (worker)
ComputeUnifiedDeviceArchitectureWorker* ComputeUnifiedDeviceArchitecture::new_subscriber() {
    ComputeUnifiedDeviceArchitectureWorker* worker = new ComputeUnifiedDeviceArchitectureWorker(this);
    __subscribers__.push_back(worker);

    // return new ComputeUnifiedDeviceArchitectureWorker(this);
    return worker;
};

void ComputeUnifiedDeviceArchitectureWorker::size_init(size_t quantity) {
    this->_size = quantity * sizeof(int);
};

// Set dimension information
template<>
void ComputeUnifiedDeviceArchitectureWorker::set_dim<void>(ComputeUnifiedDeviceArchitectureWorker::Dim* _dim) {
    this->_dim = _dim;
};

template<>
const ComputeUnifiedDeviceArchitectureWorker::Dim* ComputeUnifiedDeviceArchitectureWorker::set_dim<const ComputeUnifiedDeviceArchitectureWorker::Dim*>(ComputeUnifiedDeviceArchitectureWorker::Dim* _dim) {
    this->_dim = _dim;
    return this->_dim;
};

void ComputeUnifiedDeviceArchitectureWorker::host_init(size_t quantity) {
    for (size_t idx = 0; idx < quantity; idx++) {
        // this->_copies.emplace_back((int *)std::malloc(this->_size), this->_copies[idx].second);
        this->_copies.emplace_back((int *)std::malloc(this->_size), nullptr);
    }
};

void ComputeUnifiedDeviceArchitectureWorker::device_init(size_t quantity) {
    for (size_t idx = 0; idx < quantity; idx++) {
        hipMalloc((void **)&this->_copies[idx].second, this->_size);
    }
};

// Transform data using a callback function
void ComputeUnifiedDeviceArchitectureWorker::transform_idx(size_t idx, std::function<void(int *)> cb) {
    cb(this->_copies[idx].first);
};

void ComputeUnifiedDeviceArchitectureWorker::copy_host_to_device(size_t quantity) {
    for (size_t idx = 0; idx < quantity; idx++) {
        hipMemcpy(this->_copies[idx].second, this->_copies[idx].first, this->_size, hipMemcpyHostToDevice);
    }
};

int* ComputeUnifiedDeviceArchitectureWorker::get_host_copies_dx(size_t idx) {
    return this->_copies[idx].first;
};

int* ComputeUnifiedDeviceArchitectureWorker::get_device_copies_dx(size_t idx) {
    return this->_copies[idx].second;
};

// Swap device and host data
void ComputeUnifiedDeviceArchitectureWorker::swap_idx(size_t idx) {
    hipMemcpy(this->_copies[idx].first, this->_copies[idx].second, this->_size, hipMemcpyDeviceToHost);
};

// Clean up resources
void ComputeUnifiedDeviceArchitectureWorker::cleanup() {
    for (auto subscriber: this->_copies) {
        free(subscriber.first);
        hipFree(subscriber.second);
    }
};

// CUDA kernel for element-wise addition
__global__ void add(int *a, int *b, int *c) {
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#define N 512

// First version: Random integers without a minimum
void random_ints(int* a, int n, int max) {
    // Seed the random number generator
    static bool seeded = false;
    if (!seeded) {
        srand(static_cast<unsigned int>(time(nullptr)));
        seeded = true;
    }

    // Generate and store N random integers
    for (int i = 0; i < n; ++i) {
        a[i] = rand() % (max + 1);
    }
}

// Second version: Random integers with a minimum
void random_ints(int* a, int n, int max, int min) {
    // Seed the random number generator
    static bool seeded = false;
    if (!seeded) {
        srand(static_cast<unsigned int>(time(nullptr)));
        seeded = false;  // Reset the seed flag
    }

    // Generate and store N random integers within the specified range
    for (int i = 0; i < n; ++i) {
        a[i] = rand() % ((max - min + 1)) + min;
    }
}


int main (void) {
    ComputeUnifiedDeviceArchitecture constructor;
    ComputeUnifiedDeviceArchitectureWorker* subscriber = constructor.new_subscriber();
    subscriber->size_init(N);
    subscriber->device_init(3);
    subscriber->host_init(3);

    // Transform data using random number generation
    subscriber->transform_idx(0, [](int * _it) -> void { random_ints(_it, N, 100, 0); });
    subscriber->transform_idx(1, [](int * _it) -> void { random_ints(_it, N, 100, 0); });
    subscriber->copy_host_to_device(2);

    ComputeUnifiedDeviceArchitectureWorker::Dim* dim = new ComputeUnifiedDeviceArchitectureWorker::Dim { 
        .grid = N, .block = 1
    };
    subscriber->set_dim<void>(dim);

    // Launch add() kernel on GPU with N blocks
    add<<<N, 1>>>(
        subscriber->get_device_copies_dx(0),
        subscriber->get_device_copies_dx(1),
        subscriber->get_device_copies_dx(2)
    );

    // Copy result back to host
    subscriber->swap_idx(2);
    subscriber->cleanup();
    subscriber->get_info();
    return 0;
}
